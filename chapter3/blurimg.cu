#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include "hip/hip_runtime.h"
#include <iostream>


#define BLUR_SIZE 4
#define CHANNELS 3

__global__ void blurImgKernel(unsigned char *in, unsigned char *out, unsigned int height, unsigned int width) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int totalR = 0;
    int totalG = 0;
    int totalB = 0;
    for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE; blurRow++) {
        for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE; blurCol++) {
            int startX = x + blurRow;
            int startY = y + blurCol;
            if (startX >= 0 && startX < width && startY >= 0 && startY < height) {
                int currentIdx = (startY * width + startX) * CHANNELS;
                totalR += in[currentIdx];
                totalG += in[currentIdx + 1];
                totalB += in[currentIdx + 2];
                }
        }
    }
    int outIdx = (y * width + x) * CHANNELS;
    // Multiply by 4 because the loop runs from -BLUR_SIZE to BLUR_SIZE-1, which is BLUR_SIZE*2 on each axis
    out[outIdx] = static_cast<unsigned char>(totalR / (BLUR_SIZE * BLUR_SIZE * 4)); 
    out[outIdx + 1] = static_cast<unsigned char>(totalG / (BLUR_SIZE * BLUR_SIZE * 4));
    out[outIdx + 2] = static_cast<unsigned char>(totalB / (BLUR_SIZE * BLUR_SIZE * 4));
}

void blurImg(unsigned char *in, unsigned char *out, unsigned int height, unsigned int width) {
    unsigned char *d_in, *d_out;
    hipMalloc(&d_in, sizeof(unsigned char) * height * width * CHANNELS);
    hipMalloc(&d_out, sizeof(unsigned char) * height * width * CHANNELS);

    hipMemcpy(d_in, in, sizeof(unsigned char) * height * width * CHANNELS, hipMemcpyHostToDevice);

    dim3 threads(16, 16, 1);
    dim3 blocks(ceil(width / threads.x), ceil(height/ threads.y), 1);

    blurImgKernel<<<blocks, threads>>>(d_in, d_out, height, width);


    std::cout << "printing out" << std::endl;


    hipMemcpy(out, d_out, sizeof(unsigned char) * height * width * CHANNELS, hipMemcpyDeviceToHost);

}

int main(int argc, char** argv)
{
    // read an image
    int width, height, nChannels;

    unsigned char* data = stbi_load("doggrayscale.jpg", &width, &height, &nChannels, 0);
    
    if (data == NULL)
    {
        std::cout << "Failed to load image" << std::endl;
        return -1;
    }
    std::cout << "width: " << width << std::endl;
    std::cout << "height: " << height << std::endl;
    std::cout << "nChannels: " << nChannels << std::endl;

    unsigned char* out = new unsigned char[width * height * CHANNELS];

    std::cout << "Converting to greyscale" << std::endl;
    blurImg(data, out, height, width);

    std::cout << "Saving image. Printing raw data:" << std::endl;


    stbi_write_jpg("dogblur.jpg", width, height, CHANNELS, out, 100);
}